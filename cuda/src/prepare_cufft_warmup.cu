#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hip/hip_complex.h>
#include <hip/hip_runtime_api.h>
#include "assert.h"

typedef hipfftComplex complex;

void cufft_warper(complex *h_in, int n, int m, hipfftHandle plan, complex *h_out)
{
    const int data_size = n*m*sizeof(complex);

    // device memory allocation
    complex *d_temp;
    checkCudaErrors(hipMalloc(&d_temp,  data_size));

    // transfer data from host to device
    checkCudaErrors(hipMemcpy(d_temp, h_in, data_size, hipMemcpyHostToDevice));

	// Compute the FFT
	hipfftExecC2C(plan, d_temp, d_temp, HIPFFT_FORWARD);

    // transfer result from device to host
    checkCudaErrors(hipMemcpy(h_out, d_temp, data_size, hipMemcpyDeviceToHost));

    // cleanup
    checkCudaErrors(hipFree(d_temp));
}

void cufft_prepare(int nLevels, int *pH_N, int *pW_N,
	hipfftHandle *plan_dct_1, hipfftHandle *plan_dct_2,
	hipfftHandle *plan_dct_3, hipfftHandle *plan_dct_4,
	hipfftHandle *plan_dct_5, hipfftHandle *plan_dct_6,
	hipfftHandle *plan_dct_7, hipfftHandle *plan_dct_8)
{
	// prepare cufft plans & warmup
	printf("Preparing CuFFT plans and warmups ...  ");
	
	int Length1[1], Length2[1];
	if (nLevels >= 1)
	{
		Length1[0] = pH_N[0]; // for each FFT, the Length1 is N_height
		Length2[0] = pW_N[0];  // for each FFT, the Length2 is N_width
		hipfftPlanMany(plan_dct_1, 1, Length1,
			Length1, pW_N[0], 1,
			Length1, pW_N[0], 1,
			HIPFFT_C2C, pW_N[0]);
		hipfftPlanMany(plan_dct_2, 1, Length2,
			Length2, pH_N[0], 1,
			Length2, pH_N[0], 1,
			HIPFFT_C2C, pH_N[0]);
	}
	else
	{
		printf("No CuFFT plans prepared; out ... \n");
	}

	if (nLevels >= 2)
	{
		Length1[0] = pH_N[1]; // for each FFT, the Length1 is N_height
		Length2[0] = pW_N[1];  // for each FFT, the Length2 is N_width
		hipfftPlanMany(plan_dct_3, 1, Length1,
			Length1, pW_N[1], 1,
			Length1, pW_N[1], 1,
			HIPFFT_C2C, pW_N[1]);
		hipfftPlanMany(plan_dct_4, 1, Length2,
			Length2, pH_N[1], 1,
			Length2, pH_N[1], 1,
			HIPFFT_C2C, pH_N[1]);
	}

	if (nLevels >= 3)
	{
		Length1[0] = pH_N[2]; // for each FFT, the Length1 is N_height
		Length2[0] = pW_N[2];  // for each FFT, the Length2 is N_width				
		hipfftPlanMany(plan_dct_5, 1, Length1,
			Length1, pW_N[2], 1,
			Length1, pW_N[2], 1,
			HIPFFT_C2C, pW_N[2]);
		hipfftPlanMany(plan_dct_6, 1, Length2,
			Length2, pH_N[2], 1,
			Length2, pH_N[2], 1,
			HIPFFT_C2C, pH_N[2]);
	}

	if (nLevels >= 4)
	{
		Length1[0] = pH_N[3]; // for each FFT, the Length1 is N_height
		Length2[0] = pW_N[3];  // for each FFT, the Length2 is N_width			
		hipfftPlanMany(plan_dct_7, 1, Length2,
			Length1, pW_N[3], 1,
			Length1, pW_N[3], 1,
			HIPFFT_C2C, pW_N[3]);
		hipfftPlanMany(plan_dct_8, 1, Length2,
			Length2, pH_N[3], 1,
			Length2, pH_N[3], 1,
			HIPFFT_C2C, pH_N[3]);
	}

	// cufft warmup
	int N_width = pW_N[0];
	int N_height = pH_N[0];
	complex *h_warmup_in = new complex[N_width * N_height];
	complex *h_warmup_out = new complex[N_width * N_height];
	cufft_warper(h_warmup_in, N_width, N_height, *plan_dct_1, h_warmup_out);
	cufft_warper(h_warmup_in, N_width, N_height, *plan_dct_2, h_warmup_out);
	delete[] h_warmup_in;
	delete[] h_warmup_out;
	printf("Done.\n");
}